//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "device.h"
#include "event.h"
#include "stream.h"

namespace wp {
Stream::Stream(Device& device) : device_{device} {
    hipStream_t stream;
    check_cu(hipStreamCreateWithFlags(&stream, hipStreamDefault));
    handle_ = stream;
}

Stream::~Stream() {
    check_cu(hipStreamDestroy(static_cast<hipStream_t>(handle_)));
}

void Stream::record_event(Event& event) {
    check_cu(hipEventRecord(static_cast<hipEvent_t>(event.handle()), static_cast<hipStream_t>(handle_)));
}

void Stream::wait_event(Event& event) {
    check_cu(hipStreamWaitEvent(static_cast<hipStream_t>(handle_), static_cast<hipEvent_t>(event.handle()), 0));
}

void Stream::wait_stream(Stream& other_stream, Event& event) {
    check_cu(hipEventRecord(static_cast<hipEvent_t>(event.handle()), static_cast<hipStream_t>(other_stream.handle_)));
    check_cu(hipStreamWaitEvent(static_cast<hipStream_t>(handle_), static_cast<hipEvent_t>(event.handle()), 0));
}

void* Stream::handle() { return handle_; }

void Stream::memcpy_h2d(void* dest, void* src, size_t n) {
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice, (hipStream_t)handle_));
}
void Stream::memcpy_d2h(void* dest, void* src, size_t n) {
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost, (hipStream_t)handle_));
}
void Stream::memcpy_d2d(void* dest, void* src, size_t n) {
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice, (hipStream_t)handle_));
}

void Stream::memcpy_peer(void* dest, void* src, size_t n) {
    // NB: assumes devices involved support UVA
    check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDefault, (hipStream_t)handle_));
}

void Stream::memset(void* dest, int value, size_t n) {
    check_cuda(hipMemsetAsync(dest, value, n, (hipStream_t)handle_));
}
}  // namespace wp
