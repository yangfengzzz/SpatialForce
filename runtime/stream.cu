//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "device.h"
#include "event.h"
#include "stream.h"

namespace wp {
Stream::Stream(Device &device) : device_{device} {
    ContextGuard guard(device.get_context(), true);
    check_cu(hipStreamCreateWithFlags(static_cast<hipStream_t *>(stream), hipStreamDefault));
}

Stream::~Stream() {
    ContextGuard guard(device_.get_context(), true);
    check_cu(hipStreamDestroy(static_cast<hipStream_t>(stream)));
}

void Stream::record_event(Event &event) {
    ContextGuard guard(device_.get_context());

    check_cu(hipEventRecord(static_cast<hipEvent_t>(event.handle()), static_cast<hipStream_t>(stream)));
}

void Stream::wait_event(Event &event) {
    ContextGuard guard(device_.get_context());

    check_cu(hipStreamWaitEvent(static_cast<hipStream_t>(stream), static_cast<hipEvent_t>(event.handle()), 0));
}

void Stream::wait_stream(Stream &other_stream, Event &event) {
    ContextGuard guard(device_.get_context());

    check_cu(hipEventRecord(static_cast<hipEvent_t>(event.handle()), static_cast<hipStream_t>(other_stream.stream)));
    check_cu(hipStreamWaitEvent(static_cast<hipStream_t>(stream), static_cast<hipEvent_t>(event.handle()), 0));
}
}  // namespace wp
