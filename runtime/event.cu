//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "device.h"
#include "event.h"

namespace wp {
Event::Event(Device &device, bool enable_timing) : device_{device} {
    int flags = hipEventDefault;
    if (!enable_timing) {
        flags |= hipEventDisableTiming;
    }

    ContextGuard guard(device.get_context());
    check_cu(hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&event_), flags));
}

Event::~Event() {
    ContextGuard guard(device_.get_context(), true);
    check_cu(hipEventDestroy(static_cast<hipEvent_t>(event_)));
}

}  // namespace wp