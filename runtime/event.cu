//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "device.h"
#include "event.h"

namespace wp {
Event::Event(bool enable_timing) {
    int flags = hipEventDefault;
    if (!enable_timing) {
        flags |= hipEventDisableTiming;
    }

    check_cu(hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&event_), flags));
}

Event::~Event() { check_cu(hipEventDestroy(static_cast<hipEvent_t>(event_))); }

}  // namespace wp