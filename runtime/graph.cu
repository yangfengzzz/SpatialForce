//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "graph.h"
#include "stream.h"

namespace wp {
Graph::Graph(Stream& stream) : stream_{stream} {}

Graph::~Graph() { check_cuda(hipGraphExecDestroy((hipGraphExec_t)graph_)); }

void Graph::capture_begin() {
    check_cuda(hipStreamBeginCapture((hipStream_t)stream_.handle(), hipStreamCaptureModeGlobal));
}

void Graph::end_capture() {
    hipGraph_t graph = nullptr;
    check_cuda(hipStreamEndCapture((hipStream_t)stream_.handle(), &graph));

    if (graph) {
        // enable to create debug GraphVis visualization of graph
        // hipGraphDebugDotPrint(graph, "graph.dot", hipGraphDebugDotFlagsVerbose);

        hipGraphExec_t graph_exec = nullptr;
        // check_cuda(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

        // can use after CUDA 11.4 to permit graphs to capture hipMallocAsync() operations
        check_cuda(hipGraphInstantiateWithFlags(&graph_exec, graph, hipGraphInstantiateFlagAutoFreeOnLaunch));

        // free source graph
        check_cuda(hipGraphDestroy(graph));

        graph_ = graph_exec;
    } else {
        graph_ = nullptr;
    }
}

void Graph::launch() { check_cuda(hipGraphLaunch((hipGraphExec_t)graph_, (hipStream_t)stream_.handle())); }

}  // namespace wp