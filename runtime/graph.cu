//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"
#include "device.h"
#include "graph.h"

namespace wp {
Graph::Graph(Device &device, void *graph) : device_{device}, graph_{graph} {}

Graph::~Graph() {
    ContextGuard guard(device_.get_context());

    check_cuda(hipGraphExecDestroy((hipGraphExec_t)graph_));
}
}  // namespace wp