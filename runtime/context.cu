//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "context.h"
#include "runtime/device.h"

namespace wp {
Context::Context() {
    if (!check_cu(hipInit(0))) return;

    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        g_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                // query device info
                g_devices[i].device = device;
                g_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(g_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(hipDeviceGetAttribute(&g_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&g_devices[i].is_memory_pool_supported,
                                              hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                g_devices[i].arch = 10 * major + minor;

                hipDevice_t device;
                if (check_cu(hipDeviceGet(&device, 1))) check_cu(hipDevicePrimaryCtxRelease(device));

                g_device_map[device] = &g_devices[i];
            } else {
                return;
            }
        }
    } else {
        return;
    }
}

Device Context::creat_device() {
    active_index += 1;
    return Device(g_devices[active_index-1]);
}

}  // namespace wp