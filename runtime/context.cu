//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "context.h"

namespace wp {
Context::Context() { cuda_init(); }

int Context::cuda_init() {
    if (!check_cu(hipInit(0))) return -1;

    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        g_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                // query device info
                g_devices[i].device = device;
                g_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(g_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(hipDeviceGetAttribute(&g_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&g_devices[i].is_memory_pool_supported,
                                              hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                g_devices[i].arch = 10 * major + minor;

                g_device_map[device] = &g_devices[i];
            } else {
                return -1;
            }
        }
    } else {
        return -1;
    }

    return 0;
}

}  // namespace wp