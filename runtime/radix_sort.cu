//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include <hipcub/hipcub.hpp>

#include "cuda_util.h"
#include "device.h"
#include "radix_sort.h"

namespace wp {
std::map<void*, RadixSort::RadixSortTemp> RadixSort::g_radix_sort_temp_map;

RadixSort::RadixSort(Stream& stream) : stream_{stream} {}

void RadixSort::reserve(int n, void** mem_out, size_t* size_out) {
    hipcub::DoubleBuffer<int> d_keys;
    hipcub::DoubleBuffer<int> d_values;

    // compute temporary memory required
    size_t sort_temp_size;
    check_cuda(hipcub::DeviceRadixSort::SortPairs(nullptr, sort_temp_size, d_keys, d_values, n, 0, 32,
                                               (hipStream_t)stream_.handle()));

    RadixSortTemp& temp = g_radix_sort_temp_map[stream_.device().context()];

    if (sort_temp_size > temp.size) {
        Device::free(temp.mem);
        temp.mem = Device::alloc(sort_temp_size);
        temp.size = sort_temp_size;
    }

    if (mem_out) *mem_out = temp.mem;
    if (size_out) *size_out = temp.size;
}

void RadixSort::sort_pairs(int* keys, int* values, int n) {
    hipcub::DoubleBuffer<int> d_keys(keys, keys + n);
    hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    reserve(n, &temp.mem, &temp.size);

    // sort
    check_cuda(hipcub::DeviceRadixSort::SortPairs(temp.mem, temp.size, d_keys, d_values, n, 0, 32,
                                               (hipStream_t)stream_.handle()));

    if (d_keys.Current() != keys) stream_.memcpy_d2d(keys, d_keys.Current(), sizeof(int) * n);

    if (d_values.Current() != values) stream_.memcpy_d2d(values, d_values.Current(), sizeof(int) * n);
}
}  // namespace wp