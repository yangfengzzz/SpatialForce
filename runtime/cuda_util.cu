//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "cuda_util.h"

namespace wp {
bool check_cuda_result(hipError_t code, const char *file, int line) {
    if (code == hipSuccess) return true;

    fprintf(stderr, "Warp CUDA error %u: %s (%s:%d)\n", unsigned(code), hipGetErrorString(code), file, line);
    return false;
}

bool check_cu_result(hipError_t result, const char *file, int line) {
    if (result == hipSuccess) return true;

    const char *errString = nullptr;
    hipDrvGetErrorString(result, &errString);

    if (errString)
        fprintf(stderr, "Warp CUDA error %u: %s (%s:%d)\n", unsigned(result), errString, file, line);
    else
        fprintf(stderr, "Warp CUDA error %u (%s:%d)\n", unsigned(result), file, line);

    return false;
}

}  // namespace wp